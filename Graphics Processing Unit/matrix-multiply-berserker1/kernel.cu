
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    /*************************************************************************/
    // INSERT KERNEL CODE HERE
        
    /*************************************************************************/
    __shared__ float tilea[TILE_SIZE*TILE_SIZE];
    __shared__ float tileb[TILE_SIZE*TILE_SIZE];
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    float pvalue = 0;

    for(int p=0; p<((k-1)/TILE_SIZE + 1); p++)
    {
        if((row < m) && ((p*TILE_SIZE + threadIdx.x) < k))
        {
            tilea[threadIdx.y*TILE_SIZE+threadIdx.x] = A[row*k + p*TILE_SIZE+threadIdx.x];
        }
        else
        {
            tilea[threadIdx.y*TILE_SIZE+threadIdx.x] = 0;
        }
        if((col < n) && ((p*TILE_SIZE + threadIdx.y) < k))
        {
            tileb[threadIdx.y*TILE_SIZE+threadIdx.x] = B[col + (p*TILE_SIZE+threadIdx.y)*n];
        }
        else
        {
            tileb[threadIdx.y*TILE_SIZE+threadIdx.x] = 0;
        }
        __syncthreads();
        if((row < m) && (col < n))
        {
            for(int l=0; l<TILE_SIZE; l++)
            {
                pvalue = pvalue + tilea[threadIdx.y * TILE_SIZE + l] * tileb[l * TILE_SIZE + threadIdx.x];
            }
        }
        __syncthreads();
    }
    if ((row < m) && (col < n))
    {
        C[row*n+col] = pvalue;
    }
}

void basicSgemm(int m, int n, int k, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;
	
    /*************************************************************************/
    //INSERT CODE HERE

    /*************************************************************************/

    // Invoke CUDA kernel -----------------------------------------------------

    /*************************************************************************/
    //INSERT CODE HERE
	
    /*************************************************************************/
    dim3 DimGrid( (n - 1) / BLOCK_SIZE + 1, (m - 1) / BLOCK_SIZE + 1, 1);
    dim3 DimBlock(BLOCK_SIZE,BLOCK_SIZE,1);
    mysgemm<<<DimGrid,DimBlock>>>(m, n, k, A, B, C);
}


