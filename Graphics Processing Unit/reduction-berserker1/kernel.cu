
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512


__global__ void naiveReduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE
    // NAIVE REDUCTION IMPLEMENTATION

    __shared__ float partialsum[2 * BLOCK_SIZE];
    unsigned int tid = threadIdx.x;
    if((blockIdx.x*blockDim.x*2 + tid) < size)
    {
        // if(blockIdx.x == 976)
        // {
        //     printf("%d allocating to partialsum %d blockdim, %d blockidx, %d tid, value %f\n", blockDim.x*blockIdx.x*2 + tid, blockDim.x, blockIdx.x, tid, in[blockDim.x*blockIdx.x*2 + tid]);
        // }
        partialsum[tid] = in[blockDim.x*blockIdx.x*2 + tid];
    }
    if((blockIdx.x*blockDim.x*2 + tid + blockDim.x) < size)
    {
        // if(blockIdx.x == 976)
        // {
        //     printf("%d allocating to partialsum + blockdim, %d blockdim, %d blockidx, %d tid, value %f\n", blockDim.x*blockIdx.x*2 + tid + blockDim.x, blockDim.x, blockIdx.x, tid, in[blockDim.x*blockIdx.x*2+blockDim.x + tid]);
        // }
        partialsum[tid + blockDim.x] = in[tid + blockDim.x*blockIdx.x*2 + blockDim.x];
    }
    __syncthreads();
    unsigned int stride;
    for(stride = 1; stride <= blockDim.x; stride *= 2)
    {
        if(tid % stride == 0)
        {
            if(((2 * tid) + stride) < 2 * BLOCK_SIZE)
            {
                if((2*tid < size) && ((blockDim.x*blockIdx.x*2 + 2*tid + stride) < size))
                {
                    // if(blockIdx.x == 976)
                    // {
                    //     printf("%f 2tid %d and %f 2tid+stride %d, stride %d, corresponding numbers %d %d\n", partialsum[2*tid], 2*tid, partialsum[2*tid + stride], 2*tid + stride, stride, blockDim.x*blockIdx.x*2 + 2*tid, blockDim.x*blockIdx.x*2 + 2*tid + stride);
                    // }
                    partialsum[2*tid] += partialsum[2*tid + stride];
                }
            }
        }
        __syncthreads();
    }
    if(tid == 0)
    {
        out[blockIdx.x] = partialsum[0];
    }
    __syncthreads();

    //
    // thread block 0, it will take numbers from 0 - (2*512-1)  in
    // thread block 1 will take numbers frmo 2*512 4*512-1

}

__global__ void optimizedReduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE
    // OPTIMIZED REDUCTION IMPLEMENTATION
    __shared__ float partialsum[2 * BLOCK_SIZE];
    unsigned int tid = threadIdx.x;
    if((blockIdx.x*blockDim.x*2 + tid) < size)
    {
        // if(blockIdx.x == 976)
        // {
        //     printf("%d allocating to partialsum %d blockdim, %d blockidx, %d tid, value %f\n", blockDim.x*blockIdx.x*2 + tid, blockDim.x, blockIdx.x, tid, in[blockDim.x*blockIdx.x*2 + tid]);
        // }
        partialsum[tid] = in[blockDim.x*blockIdx.x*2 + tid];
    }
    if((blockIdx.x*blockDim.x*2 + tid + blockDim.x) < size)
    {
        // if(blockIdx.x == 976)
        // {
        //     printf("%d allocating to partialsum + blockdim, %d blockdim, %d blockidx, %d tid, value %f\n", blockDim.x*blockIdx.x*2 + tid + blockDim.x, blockDim.x, blockIdx.x, tid, in[blockDim.x*blockIdx.x*2+blockDim.x + tid]);
        // }
        partialsum[tid + blockDim.x] = in[tid + blockDim.x*blockIdx.x*2 + blockDim.x];
    }
    __syncthreads();
    unsigned int stride;
    for(stride=blockDim.x; stride>0; stride = stride/2)
    {
        if((tid < stride) && (blockDim.x*blockIdx.x*2 + tid + stride < size))
        {
            partialsum[tid] += partialsum[tid + stride];
        }
        __syncthreads();
    }
    if(tid == 0)
    {
        out[blockIdx.x] = partialsum[0];
    }

}
