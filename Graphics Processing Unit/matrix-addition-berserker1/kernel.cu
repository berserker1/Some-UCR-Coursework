
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void matAdd(int dim, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (dim x dim) matrix
     *   where B is a (dim x dim) matrix
     *   where C is a (dim x dim) matrix
     *
     ********************************************************************/

    /*************************************************************************/
    // INSERT KERNEL CODE HERE
    // printf("%f %f Row 25-0 col 0\n", A_h[250000], B_h[250000]);
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("C %f A %f B %f\n", C[250000], A[250000], B[250000]);
    if ((Row < dim) && (Col < dim))
    {
        C[Row*dim + Col] = A[Row*dim + Col] + B[Row*dim + Col];
        // if(C[Row*dim + Col] != (A[Row*dim + Col] + B[Row*dim + Col]))
        // {
        //     printf("C %f A %f B %f\n", C[Row*dim + Col], A[Row*dim + Col], B[Row*dim + Col]);
        //     return;    
        // }
        // printf("C %f A %f B %f  Row %d Col %d\n", C[Row*dim + Col], A[Row*dim + Col], B[Row*dim + Col], Row, Col);
    }
    /*************************************************************************/

}

void basicMatAdd(int dim, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;
	
    /*************************************************************************/
    //INSERT CODE HERE
    const unsigned int grid_size = (dim - 1)/BLOCK_SIZE + 1;
    dim3 dimgrid(grid_size, grid_size, 1);
    dim3 dimblock(BLOCK_SIZE, BLOCK_SIZE, 1);
    // printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n", dimgrid.x, dimgrid.y, dimgrid.z, dimblock.x, dimblock.y, dimblock.z);
    // printf("%f %f Row 25-0 col 0\n", A[999], B[999]);
    /*************************************************************************/
	
	// Invoke CUDA kernel -----------------------------------------------------
    matAdd<<<dimgrid, dimblock>>>(dim, A, B, C);
    /*************************************************************************/
    //INSERT CODE HERE
    /*************************************************************************/

}

